
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 200
#define THREADS_PER_BLOCK 512

__global__ void dot(int *a, int *b, int *c) {
    __shared__ int temp[THREADS_PER_BLOCK];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    temp[threadIdx.x] = a[index] * b[index];

    __syncthreads();
    
    if( 0 == threadIdx.x) {
        int sum = 0;
        for(int i = 0; i < THREADS_PER_BLOCK; i++)
            sum += temp[i];
        atomicAdd(c, sum);
    }
}

int main(void){
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;
    int size = N * sizeof(int);

    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, sizeof(int));

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)calloc(sizeof(int),0);

    for(int i = 0; i < N; i++){
        a[i] = 1;
        b[i] = 1;
    }

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

    dot<<< N/THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("C: %d\n",*c);

    free(a); free(b); free(c);
    hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
    return 0;
}