#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "device_atomic_functions.h"
#include <stdio.h>
#include <stdlib.h>
#define N 2048
#define THREADS_PER_BLOCK 512

__global__ void dot(int *a, int *b, int *c)
{
    __shared__ int temp[THREADS_PER_BLOCK];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    temp[threadIdx.x] = a[index] * b[index];

    __syncthreads();

    if (threadIdx.x == 0)
    {
        int sum = 0;
        for (int i = 0; i < THREADS_PER_BLOCK; i++)
        {
            sum += temp[i];
        }
        atomicAdd(c, sum);
    }
}

int main()
{
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;
    int size = N * sizeof(int);

   //allocate space for the variables on the device
    hipMalloc((void **)&dev_a, size);
    hipMalloc((void **)&dev_b, size);
    hipMalloc((void **)&dev_c, sizeof(int));

   //allocate space for the variables on the host
   a = (int *)malloc(size);
   b = (int *)malloc(size);
   c = (int *)malloc(sizeof(int));

   //this is our ground truth
   int sumTest = 0;
   //generate numbers
   for (int i = 0; i < N; i++)
   {
       a[i] = 1;
       b[i] = 1;
       sumTest += a[i] * b[i];
   }

   *c = 0;

   hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
   hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
   hipMemcpy(dev_c, c, sizeof(int), hipMemcpyHostToDevice);

   dot<<< N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(dev_a, dev_b,    dev_c);

   hipMemcpy(c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

   printf("%d ", *c);
   printf("%d ", sumTest);

   free(a);
   free(b);
   free(c);

   hipFree(a);
   hipFree(b);
   hipFree(c);


   return 0;

 }