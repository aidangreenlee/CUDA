
#include <hip/hip_runtime.h>
#include <iostream>
#define THREADS_PER_BLOCK 512
#define N 2048*2048


__global__ void dot(int *a, int *b, int *c){
    __shared__ int temp[THREADS_PER_BLOCK];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    temp[threadIdx.x] = a[index] * b[index];

    __syncthreads();

    if( threadIdx.x == 0){
        int sum = 0;
        for(int i = 0; i < THREADS_PER_BLOCK; i++){
            sum += temp[i];
        }
        atomicAdd(c, sum);
    }
}

int main(){
    int *a, *b;
    int *c, *dev_c;
    int *dev_a, *dev_b;
    int test = 0.0f;
    long long size = N * sizeof(int);

    hipMalloc((void **)&dev_a, size);
    hipMalloc((void **)&dev_b, size);
    hipMalloc((void **)&dev_c, sizeof(int));

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(sizeof(int));

    *c = 0;

    for(int i = 0; i < N; i++){
        a[i] = 1;
        b[i] = 1;
        test += a[i] * b[i];
    }

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, sizeof(int), hipMemcpyHostToDevice);

    dot<<< N/THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    
    std::cout << "dot product: " << *c << std::endl;
    std::cout << "test: " << test << std::endl;

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    free(a);
    free(b);
    free(c);
    return 0;
}